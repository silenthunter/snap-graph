#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__device__ const int MAX_DEGREE = 4;

const int BLOCK_WIDTH = 8;
const int BLOCK_HEIGHT = 8;

__device__ void sortEdges(int* edges, int* sorted)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int i = x + y * gridDim.x * blockDim.x;
	int n1 = edges[i * 2];
	int n2 = edges[i * 2 + 1];
	
	int* arrStart = &sorted[n1 * MAX_DEGREE];
	int retnVal = 1;
	while(retnVal != 0)
		retnVal = atomicCAS(arrStart, 0, n2);
}


//HACK: This will be incredibly slow  on CUDA!
__device__ int findNext(int* edges, int numEdge, int v, int* destination)
{
	int count = 0;

	for(int i = 0; i < numEdge * 2; i+=2)
	{
		if(edges[i] == v)
			destination[count++] = edges[i + 1];
	}

	return count;
}

__device__ void pushQueue(int element, int* queue, int queueSize, int* head, int* tail)
{
	queue[*tail] = element;
	*tail = (*tail + 1) % queueSize;
}

__device__ int popQueue(int* queue, int queueSize, int* head, int* tail)
{
	int retn = queue[*head];
	*head = (*head + 1) % queueSize;
	
	return retn;
}

__device__ void pushStack(int element, int* stack, int* head)
{
	stack[*head] = element;
	*head = *head + 1;
}

__device__ int popStack(int* stack, int* head)
{
	*head = *head - 1;
	int retn = stack[*head];
	
	return retn;
}

const int ELEMENTS = 512;
const int S_SIZE = ELEMENTS;
const int P_SIZE = ELEMENTS;
const int PATH_SIZE = ELEMENTS;
const int D_SIZE = ELEMENTS;
const int Q_SIZE = ELEMENTS;


__device__ void doAlg(int numVert, int* edges, int numEdges, float* BC, int* glob, float* globDep)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.y + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	int PTR_OFFSET = idx * (S_SIZE + (P_SIZE * MAX_DEGREE) + D_SIZE + Q_SIZE + PATH_SIZE);
	
	int* S = &glob[PTR_OFFSET];
	int S_head = 0;
	PTR_OFFSET += S_SIZE;
	
	int* P = &glob[PTR_OFFSET];
	//Blank the previous items
	for(int i = 0; i < P_SIZE; i++)
		for(int j = 0; j < MAX_DEGREE; j++)
		{
			P[i + P_SIZE * j] = -1;
		}
	PTR_OFFSET += P_SIZE * MAX_DEGREE;

	int* pathCount = &glob[PTR_OFFSET];
	for(int i = 0; i < PATH_SIZE; i++)
	{
		pathCount[i] = 0;
	}
	pathCount[idx] = 1;
	PTR_OFFSET += PATH_SIZE;

	int* d = &glob[PTR_OFFSET];
	for(int i = 0; i < D_SIZE; i++)
	{
		d[i] = -1;
	}
	d[idx] = 0;
	PTR_OFFSET += D_SIZE;
	
	int* Q = &glob[PTR_OFFSET];
	int Q_head = 0;
	int Q_tail = 0;
	PTR_OFFSET += Q_SIZE;
	
	pushQueue(idx, Q, Q_SIZE, &Q_head, &Q_tail);

	while(Q_head != Q_tail)
	{
		int v = popQueue(Q, Q_SIZE, &Q_head, &Q_tail);
		pushStack(v, S, &S_head);

		int w[MAX_DEGREE];
		int edgeCount = findNext(edges, numEdges, v, w);

		for(int i = 0; i < edgeCount; i++)
		{
			int wNode = w[i];
			if(d[wNode] < 0)
			{
				pushQueue(wNode, Q, Q_SIZE, &Q_head, &Q_tail);
				d[wNode] = d[v] + 1;
			}
			
			if(d[wNode] == d[v] + 1)
			{
				pathCount[wNode] = pathCount[wNode] + pathCount[v];
				
				//Append v to the PrevNode list
				for(int j = 0; j < MAX_DEGREE; j++)
				{
					if(P[wNode + P_SIZE * j] < 0)
					{
						P[wNode + P_SIZE * j] = v;
						break;
					}
				}
			}
		}

	}
	
	float* dep = &globDep[idx * ELEMENTS];
	
	while(S_head > 0)
	{
		int w = popStack(S, &S_head);
		
		//Loop through each v in P[w]
		for(int i = 0; i < MAX_DEGREE; i++)
		{
			int v = P[w + P_SIZE * i];
			if(v < 0) continue;

			dep[v] = dep[v] + ((float)pathCount[v]/(float)pathCount[w]) * (1 + dep[w]);
		}
		
		if(w != idx)
		{
			atomicAdd(&BC[w], dep[w]);
		}
	}
	
}

__global__ void betweennessCentrality(int numVert, int numEdges, int *edges, float* BC, int* glob, float* dep)
{
	extern __shared__ int path[];
	
	//sortEdges(edges, path);
	int x = blockDim.x * blockIdx.x + threadIdx.x;	
	int y = blockDim.y * blockIdx.x + threadIdx.y;	
	int idx = x + y * blockDim.x * gridDim.x;

	BC[idx] = 0.0f;

	__syncthreads();

	doAlg(numVert, edges, numEdges, BC, glob, dep);

		
}

void cuda_bc()
{
}

/*int main()
{
	const int elements = ELEMENTS;

	//cudaProfilerStart();
	int *d_mem;
	int *h_edge;
	int *d_edge;
	float *d_bc;
	float *h_bc;
	int *d_glob;
	float *d_dep;
	
	cudaMalloc((void**)&d_mem, sizeof(int) * elements);
	
	h_edge = (int*)malloc(sizeof(int) * elements * 2);
	cudaMalloc((void**)&d_edge, sizeof(int) * elements * 2);

	h_bc = (float*)malloc(sizeof(float) * elements);
	cudaMalloc((void**)&d_bc, sizeof(float) * elements);

	cudaMalloc((void**)&d_glob, sizeof(int) * elements * elements * 8);
	cudaMalloc((void**)&d_dep, sizeof(float) * elements * elements);
	
	//Init edges
	for(int i = 0; i < elements - 1; i++)
	{
		h_edge[i * 2] = i % elements;
		h_edge[i * 2 + 1] = (i + 1) % elements;
	}
	cudaMemcpy(d_edge, h_edge, sizeof(int) * elements * 2, cudaMemcpyHostToDevice);
	
	dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 grid(elements / (BLOCK_WIDTH * BLOCK_HEIGHT));
	//test<<<grid,block>>>(d_mem);
	betweennessCentrality<<<grid,block>>>(elements, elements - 1, d_edge, d_bc, d_glob, d_dep);
	cudaError_t error = cudaGetLastError();
	
	int* h_mem = (int*)malloc(sizeof(int) * elements);
	cudaMemcpy(h_mem, d_mem, sizeof(int) * elements, cudaMemcpyDeviceToHost);
	cudaMemcpy(h_bc, d_bc, sizeof(float) * elements, cudaMemcpyDeviceToHost);
	

	for(int i = 0; i < elements; i++)
	{
		cout << h_bc[i] << endl;
	}
	//cout<<elements<<endl;
	
	//cudaProfilerStop();
	
	cudaDeviceReset();
	cout << cudaGetErrorString(error) << endl;
	
	return 0;
}*/
